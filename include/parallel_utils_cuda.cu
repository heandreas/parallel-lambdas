#include "hip/hip_runtime.h"
#pragma once

#include <vector>
#include <type_traits>
#include <algorithm>
#include <thrust/device_vector.h>
#include <hip/hip_runtime.h>

template<typename T, typename F>
__global__  void mapKernel(int n, T* pThreadData, F mapper)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	pThreadData += index;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += stride)
		mapper(i, *pThreadData);
}

class ParallelUtilsCuda
{
public:
	template<typename Int>
	static Int ceilDivision(Int a, Int b)
	{
		return (a + b - 1) / b;
	}

	template<class OpListType>
	class PerThreadOperation
	{
	public:
		virtual ~PerThreadOperation() {}
		virtual int doWork(OpListType* pOpList) = 0;
	};

	struct ScheduleHints
	{
		bool m_UseDynamicScheduling = false;
		size_t m_DynamicSchedulingChunkSize = 1;
		ScheduleHints(bool useDynamicScheduling = false, size_t dynamicSchedulingChunkSize = 1) : m_UseDynamicScheduling(useDynamicScheduling), m_DynamicSchedulingChunkSize(dynamicSchedulingChunkSize) {}
		static ScheduleHints dynamic(size_t dynamicSchedulingChunkSize = 1) { return ScheduleHints(true, dynamicSchedulingChunkSize); }
	};

	/*template<class PerThreadData, class FoldFunc>
	class FoldOp : public PerThreadOperation<PerThreadData>
	{
		FoldFunc m_FoldFunc;
		bool m_Ordered = false;
	public:
		virtual int doWork(int numThreads, int, PerThreadData& perThreadData) override
		{
			return 0;
		}

	public:
		FoldOp(const FoldFunc& func, bool ordered = false) : m_FoldFunc(func), m_Ordered(ordered) {}
	};*/

	template<class CreateData>
	using ReturnOfCreateData = typename std::result_of<CreateData(int)>::type;

	template<class CreateThreadLocalDataFunc>
	class Team
	{
		using ThreadLocalData = ReturnOfCreateData<CreateThreadLocalDataFunc>;
		using SelfType = Team<CreateThreadLocalDataFunc>;
		CreateThreadLocalDataFunc m_CreateThreadLocalDataFunc;
		int m_NumThreads;
		thrust::device_vector<ThreadLocalData> m_PerThreadData;

		template<class Mapper>
		Team* _mapWithLocalData(size_t numElements, const Mapper& mapper, const ScheduleHints& scheduleHints = ScheduleHints())
		{
			int blockSize = 256;
			int numBlocks = std::min(m_NumThreads, (int)ceilDivision((int)numElements, blockSize));
			mapKernel << <numBlocks, blockSize >> > ((int)numElements, thrust::raw_pointer_cast(&m_PerThreadData[0]), mapper);
			hipDeviceSynchronize();
			return this;
		}

	public:
		Team(const CreateThreadLocalDataFunc& func, int numThreads) : m_CreateThreadLocalDataFunc(func), m_NumThreads(numThreads) {}

		//! Performs a parallel map operation on the range [0, numElements). The mapper should take an index and the thread-local data as argument.
		template<class Mapper>
		Team* mapWithLocalData(size_t numElements, const Mapper& mapper, const ScheduleHints& scheduleHints = ScheduleHints())
		{
			if (m_PerThreadData.empty()) m_PerThreadData.resize(m_NumThreads);
			return _mapWithLocalData(numElements, mapper, scheduleHints);
		}
		//! Performs a parallel map operation on the range [0, numElements). The mapper should take an index as argument.
		template<class Mapper>
		Team* map(size_t numElements, const Mapper& mapper, const ScheduleHints& scheduleHints = ScheduleHints())
		{
			return _mapWithLocalData(numElements, [mapper] __device__ (int i, ThreadLocalData&) { mapper(i); }, scheduleHints);
		}
		//! Calls the given lambda synchronized for each thread, passing the thread-local data. Intended for fold / reduce operations.
		/*template<class FoldFunc>
		OpList* fold(const FoldFunc& foldFunc, bool ordered = false)
		{
			m_List.emplace_back(new FoldOp<ThreadLocalData, FoldFunc>(foldFunc, ordered));
			return this;
		}*/

		//! Adds a user-defined operation to the OpList. The CustomAdder class must implement a static function addOp(OpList*, params...).
		template<typename CustomAdder, typename... Params>
		Team* customOp(const Params& ... params)
		{
			CustomAdder::addOp(this, params...);
			return this;
		}
	};

	struct NoThreadLocalData { int operator()(int) const { return 0; } };

	//! @param createThreadLocalDataFunc is a function that initializes and returns the thread-local data passed to the mappers. The thread index is passed as an int parameter to the createData function.
	//! @param numThreads is the number of threads that should be used for the computation.
	template<class CreateThreadLocalDataFunc = NoThreadLocalData>
	static std::unique_ptr<Team<CreateThreadLocalDataFunc>> createTeam(const CreateThreadLocalDataFunc& createThreadLocalDataFunc = NoThreadLocalData(), int numThreads = 1 << 15)
	{
		return std::make_unique<Team<CreateThreadLocalDataFunc>>(createThreadLocalDataFunc, numThreads);
	}
};
